#include "hip/hip_runtime.h"

#include "cu_mult_handle.h"
#include "cu_crt.h"
#include "cu_icrt.h"
#include <hip/hip_cooperative_groups.h>
#include <hip/hip_runtime.h> 
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <chrono>
__constant__  long c_logN[1];

#define CUDA_API_PER_THREAD_DEFAULT_STEAM
#ifndef MAX_COEFF_SIZE
    #define MAX_COEFF_SIZE 64
#endif

std::chrono::high_resolution_clock::time_point n1, n2;
#define START() n1 = std::chrono::high_resolution_clock::now();
#define END() n2 = std::chrono::high_resolution_clock::now();
#define SUM(t) t += (double)std::chrono::duration_cast<std::hrono::nanoseconds >(n2-n1).count() ;
#define PRINTTIME(msg) std::cout << msg << " time = " << (double)std::chrono::duration_cast<std::chrono::nanoseconds >(n2 - n1).count() / 1000 << " microseconds" << std::endl;





__global__ void mulModBarrett_kernel(uint64_t* r,uint64_t* a,uint64_t* b, uint64_t* pVec, uint64_t* prVec, long* pTwok)
{
    uint64_t N = 1<<c_logN[0];
    int idx = blockIdx.y *blockDim.x + threadIdx.x;
	//unsigned __int128 mul = static_cast<unsigned __int128>(a) * b;
    uint64_t p = pVec[blockIdx.x];
    uint64_t pr = prVec[blockIdx.x];
    uint64_t twok = pTwok[blockIdx.x];
	uint64_t abot = a[blockIdx.x*N +idx ] * b[blockIdx.x*N +idx ];
	uint64_t atop = __umul64hi(a[blockIdx.x*N +idx ],b[blockIdx.x*N +idx ]);
    uint64_t tmp_high = 0;
    uint64_t tmp_low = __umul64hi(abot,pr);
    
    uint64_t tmp_atop_pr =  atop * pr;
    tmp_low += tmp_atop_pr;
    tmp_high = __umul64hi(atop,pr) + (tmp_low <tmp_atop_pr);
    tmp_low =  (tmp_low >> (twok - 64)) | (tmp_high << (128 -twok)) ;
    tmp_high >>= twok - 64;
	tmp_low =  tmp_low * p;
    tmp_high = tmp_high * p + __umul64hi(tmp_low,p);

    tmp_low = abot - tmp_low;
    r[blockIdx.x*N +idx] = tmp_low;
	if(r[blockIdx.x*N +idx ] >= p) {
		r[blockIdx.x*N +idx] -= p;
	}
}

__device__ void cuda_NTT_inner(uint64_t* a,long t, uint64_t W, uint64_t p,uint64_t pInv)
{
    uint64_t T = a[t];
    uint64_t U0 =  T* W;
    uint64_t U1 =  __umul64hi(T, W);
    uint64_t Q = U0 * pInv;
    uint64_t H =  __umul64hi(Q, p);
    uint64_t V = U1 < H ? U1 + p - H : U1 - H;
    a[t] = a[0] < V ? a[0] + p - V : a[0] - V;
    a[0] += V;
    if (a[0] > p)
        a[0] -= p;

}

__global__ void cuda_NTT_kernel(uint64_t* r, uint64_t* scaledRootInvPows, uint64_t* pVec, uint64_t* pInvVec) {
    cooperative_groups::grid_group grp = cooperative_groups::this_grid();
    long N = 1 << c_logN[0];
    long t = N;
     
    //printf("%ld\n",blockIdx.x << c_logN[0]);
    int idx = blockIdx.y *blockDim.x + threadIdx.x;
    for(long m=1;m<N;m<<=1)
    {
        t >>=1;
        int j = idx + t*(idx/t);
        uint64_t W = scaledRootInvPows[m + (idx/t)];

        cuda_NTT_inner(r+j,t,W,pVec[0],pInvVec[0]);
        grp.sync();
    }


}

__device__ void cuda_INTT_inner(uint64_t* a,long t, uint64_t W, uint64_t p,uint64_t pInv)
{
    uint64_t U = a[0] + a[t];
    if (U > p)
        U -= p;
    uint64_t T =a[0] < a[t] ? a[0] + p - a[t] : a[0] - a[t];
    uint64_t U0 =  T* W;
    uint64_t U1 =  __umul64hi(T, W);
    uint64_t Q = U0 * pInv;
    uint64_t H =  __umul64hi(Q, p);
    a[0] = U;
    a[t] = (U1 < H) ? U1 + p - H : U1 - H;


}

__device__ void cuda_INTT_scaler(uint64_t* a,long t, uint64_t NScale, uint64_t p,uint64_t pInv)
{
    uint64_t T = a[0];
    uint64_t U0 = T * NScale;
    uint64_t U1 =  __umul64hi(T, NScale);
    uint64_t Q = U0 * pInv;
    uint64_t H =__umul64hi(Q, p);
    a[0] = (U1 < H) ? U1 + p - H : U1 - H;

     T = a[t];
     U0 = T * NScale;
     U1 =  __umul64hi(T, NScale);
     Q = U0 * pInv;
     H = __umul64hi(Q, p);
    a[t] = (U1 < H) ? U1 + p - H : U1 - H;
}

__global__ void cuda_INTT_kernel(uint64_t* r, uint64_t* scaledRootPows, uint64_t* scaledNInv , uint64_t* pVec, uint64_t* pInvVec) 
{
    cooperative_groups::grid_group grp = cooperative_groups::this_grid();
    long N = 1 << c_logN[0];
    long t = 1;
     
    //printf("%ld\n",blockIdx.x << c_logN[0]);
    int idx = blockIdx.y *blockDim.x + threadIdx.x;

	for (long m=N; m > 1; m >>= 1) {
        long h = m >> 1;
        int j = idx + t*(idx/t);
        uint64_t W = scaledRootPows[ h + (idx/(t))];

        cuda_INTT_inner(r+j,t,W,pVec[0],pInvVec[0]);
        t <<= 1;
        
        grp.sync();
	}

    cuda_INTT_scaler(r+ idx,N>>1,scaledNInv[0],pVec[0],pInvVec[0]);

}

__global__ void test_print(uint64_t& test)
{
    printf("test: %lu  \n",test);
}

cuda_mult_handler::cuda_mult_handler(long nprimes_,long logN_)
{
    nprimes = nprimes_;
    logN = logN_;
    N = 1 << logN;
    coeff_size = nprimes *sizeof(uint64_t);

    crt_coeff_size  = coeff_size * N;
    hipStreamCreate(&stream_mem);
    

    rem_builder = new cu_remainder_build(N,nprimes);
    icrt_builder = new cuda_reconstruct(N,nprimes);

}

cuda_mult_handler::~cuda_mult_handler()
{
    hipFree(c_logN);
    // param for NTT
    hipFree( c_pVec);
    hipFree( c_prVec);
    hipFree( c_pTwok);
    hipFree( c_pInvVec);
    hipFree( c_scaledRootPows);
    hipFree( c_scaledRootInvPows);
    hipFree( c_scaledNInv);
    hipFree( c_logN);


    hipStreamDestroy(stream_mem);

}

void cuda_mult_handler::ntt_memAlloc(long npimes_,long logN_)
{

    hipMalloc(&c_pVec ,coeff_size);
    hipMalloc(&c_prVec,coeff_size);
    hipMalloc(&c_pInvVec,coeff_size);
    hipMalloc(&c_pTwok,nprimes *sizeof(long));
    hipMalloc(&c_scaledRootPows, coeff_size * N);
    hipMalloc(&c_scaledRootInvPows, coeff_size * N);
    hipMalloc(&c_scaledNInv, coeff_size);

    //(*icrt_builder).memAlloc();

    (*icrt_builder).memAlloc(nprimes);

}

void cuda_mult_handler::param_cpy(uint64_t* pVec, uint64_t* prVec, uint64_t* pInvVec, long* pTwok, uint64_t** scaledRootPows,uint64_t** scaledRootInvPows, uint64_t* scaledNInv, NTL::ZZ** pHat, uint64_t** pHatInvModp, unsigned long** coeffpinv_array, NTL::ZZ* pProd,NTL::ZZ* pProdh)
{
    
    hipMemcpyAsync(c_pVec,pVec,coeff_size,hipMemcpyHostToDevice,stream_mem);
    hipMemcpyAsync(c_prVec,prVec,coeff_size,hipMemcpyHostToDevice,stream_mem);
    hipMemcpyAsync(c_pInvVec,pInvVec,coeff_size,hipMemcpyHostToDevice,stream_mem);
    hipMemcpyAsync(c_pTwok,pTwok,nprimes *sizeof(long),hipMemcpyHostToDevice,stream_mem);
    for(long i=0; i <nprimes;i ++)
    {
        hipMemcpyAsync(c_scaledRootPows + N*i,scaledRootPows[i], N*sizeof(uint64_t),hipMemcpyHostToDevice,stream_mem);
        hipMemcpyAsync(c_scaledRootInvPows + N*i,scaledRootInvPows[i], N*sizeof(uint64_t),hipMemcpyHostToDevice,stream_mem);
    }

    hipMemcpyAsync(c_scaledNInv,scaledNInv,coeff_size,hipMemcpyHostToDevice,stream_mem);
    hipMemcpyToSymbolAsync(HIP_SYMBOL(c_logN),&logN,sizeof(long),0,hipMemcpyHostToDevice,stream_mem);
    (*icrt_builder).c_pVec = c_pVec;

    (*icrt_builder).make_param(nprimes, pHat, pHatInvModp,coeffpinv_array, pProd, pProdh);
}

void cuda_mult_handler::param_position_set()
{
    (*icrt_builder).np = np;
    (*icrt_builder).c_pHatnp = (*icrt_builder).c_pHat + ((np-1)*np/2)* 64;
    (*icrt_builder).c_pHatInvModpnp = (*icrt_builder).c_pHatInvModp + (np-1)*np/2;
    (*icrt_builder).c_coeffpinv_arraynp = (*icrt_builder).c_coeffpinv_array + (np-1)*np/2;
    (*icrt_builder).c_pProdnp = (*icrt_builder).c_pProd + np-1;
    (*icrt_builder).c_pProdhnp = (*icrt_builder).c_pProdh + np-1;
}

uint64_t* cuda_mult_handler::host_uint_to_cuda_crt(uint64_signed x, hipStream_t stream)
{

    if( !(*rem_builder).is_table_maked )
    {
        c_tbl = (*rem_builder).cuda_tbl_build(c_pVec,stream);
        (*rem_builder).is_table_maked = true;
    }
        



    uint64_t* c_data;// = mem_split_flag ? c_data_b : c_data_a;
    bool* c_neg_check;// = mem_split_flag ? c_neg_check_b : c_neg_check_a;

    hipMalloc(&c_data,N*MAX_COEFF_SIZE*sizeof(uint64_t));
    hipMalloc(&c_neg_check,N*sizeof(bool));
    //mem_split_flag = mem_split_flag ? false:true;
    hipMemcpyAsync( c_data, x.data,N* MAX_COEFF_SIZE * sizeof(uint64_t),hipMemcpyHostToDevice,stream);  
    hipMemcpyAsync( c_neg_check, x.neg_check,N*sizeof(bool),hipMemcpyHostToDevice,stream);

    

    uint64_t *c_result = (*rem_builder).host_uint_to_cuda_remainder(c_data, c_neg_check, c_pVec, c_tbl,stream);
    hipFree(c_data);
    hipFree(c_neg_check);
    return c_result;
}

void cuda_mult_handler::ntt_poly_cpy(uint64_t* ra,uint64_t* rb)
{
    //hipMemcpyAsync(c_ra,ra,crt_coeff_size,hipMemcpyHostToDevice,stream_mem);
    //hipMemcpyAsync(c_rb,rb,crt_coeff_size,hipMemcpyHostToDevice,stream_mem);

    //c_ra = ra;
    //c_rb= rb;

}

uint64_t* cuda_mult_handler::ntt_to_host(uint64_t* c_x)
{

    uint64_t* x = new uint64_t[np << logN]();
    hipMemcpyAsync(x, c_x,(np << logN)*sizeof(uint64_t),hipMemcpyDeviceToHost,stream_mem);
    

    return x;
    //ntt_memFree();
}

void cuda_mult_handler::cuda_NTT_run(uint64_t* c_poly_ring,hipStream_t stream, int thread_max )
{
    //hipDeviceSynchronize(); 
    int block_max =0;
    if (logN-1 < thread_max)
    {
        thread_max = 1<<(logN-1);
        block_max = 1;
    }
    else
    {
        block_max = 1 << (logN-thread_max-1);
        thread_max =1 << thread_max;
    }
    dim3 grids(1,block_max,1);
    dim3 threads(thread_max,1,1);

    //multi_stream process
    long* m; hipMalloc(&m, sizeof(long));
    hipMemsetAsync(m,0,sizeof(long),stream);
    for(long i =0; i < np; i ++)
    {
        uint64_t* c_poly_ring_split = c_poly_ring + N*i;
        uint64_t* c_scaledRootPows_split = c_scaledRootPows + N*i;
        uint64_t* c_p = c_pVec + i;
        uint64_t* c_pInv = c_pInvVec +i;
        void* arguments[] = {(void*)&(c_poly_ring_split),  (void*)&c_scaledRootPows_split,(void*)& c_p, (void*)&c_pInv};
        
        
        //cout<< "NTT " <<hipGetErrorString(
            hipLaunchCooperativeKernel((void*) cuda_NTT_kernel, grids, threads, arguments,0,stream)
        ;//) <<endl;
        //hipStreamDestroy(stream);
    }        
}

void cuda_mult_handler::cuda_INTT_run(uint64_t* c_poly_ring,int thread_max )
{
    //hipDeviceSynchronize(); 
    int block_max =0;
    if (logN-1 < thread_max)
    {
        thread_max = 1<<(logN-1);
        block_max = 1;
    }
    else
    {
        block_max = 1 << (logN-thread_max-1);
        thread_max =1 << thread_max;
    }
    dim3 grids(1,block_max,1);
    dim3 threads(thread_max,1,1);

    //multi_stream process
    for(long i =0; i < np; i ++)
    {
        // hipStream_t stream;
        // hipStreamCreate(&stream);
        uint64_t* c_poly_ring_split = c_poly_ring + N*i;
        uint64_t* c_scaledRootInvPows_split = c_scaledRootInvPows + N*i;
        uint64_t* c_p = c_pVec + i;
        uint64_t* c_pInv = c_pInvVec +i;
        uint64_t* c_scaled = c_scaledNInv +i;
        void* arguments[] = {(void*)&(c_poly_ring_split), (void*)&c_scaledRootInvPows_split, (void*)&c_scaled,(void*)& c_p, (void*)&c_pInv};
        
        //cout <<hipGetErrorString(
            hipLaunchCooperativeKernel((void*) cuda_INTT_kernel, grids, threads, arguments,0)
        ;//) <<endl;
        //hipStreamDestroy(stream);
    }        
}

uint64_t* cuda_mult_handler::mulModBarrett_cuda(uint64_t* c_ra,uint64_t* c_rb ,int thread_max)
{

    uint64_t* c_rx;
    hipMalloc(&c_rx,crt_coeff_size);

    int block_max =0;
    if (logN < thread_max)
    {
        thread_max = 1<<(logN);
        block_max = 1;
    }
    else
    {
        block_max = 1 << (logN-thread_max);
        thread_max =1 << thread_max;
    }
    dim3 grids(np,block_max,1);
    dim3 threads(thread_max,1,1);
    mulModBarrett_kernel<<<grids,threads>>>(c_rx,c_ra,c_rb,c_pVec,c_prVec,c_pTwok );

    return c_rx;
}

void cuda_mult_handler::cuda_uint_to_host_icrt(NTL::ZZ* x,uint64_t* c_rx ,const NTL::ZZ& mod)
{
    (*icrt_builder).cuda_icrt_run(x,c_rx,mod);
}

uint64_signed cuda_mult_handler::zz_to_uint(NTL::ZZ* data)
{
    uint64_t* data_tmp = new uint64_t[MAX_COEFF_SIZE * N]{0,};
    bool* neg_check = new bool[N];
    //
    for(long j =0;j<N;j++)
    {
      NTL::ZZ a_data = data[j];
      long size = a_data.size();
      if(size == 0 ) size = 1;
      
      if (a_data < 0)
      {
        a_data = -a_data;
        neg_check[j] = true;
      }
      else{
        neg_check[j] = false;
      }

      //uint64_t* tmp = new uint64_t[size]{0,};
      for(long i =0;i < size ; i++)
      {
        data_tmp[i + j*MAX_COEFF_SIZE] = NTL::conv<uint64_t>(a_data);
        a_data >>= 64;
      }
      //hipMemcpy( c_data+j*MAX_COEFF_SIZE, tmp,(size)*sizeof(uint64_t),hipMemcpyHostToDevice);
    }
    uint64_signed uint64_data;
    uint64_data.data = data_tmp;
    uint64_data.neg_check = neg_check;

    //delete[] data_tmp;
    //delete[] neg_check;
    return uint64_data;
}

uint64_t* cuda_mult_handler::host_ntt_to_cuda_ntt(uint64_t* ra,hipStream_t stream_mem)
{
    uint64_t* result;
    hipMalloc(&result,np*N*sizeof(uint64_t));
    hipMemcpyAsync(result, ra, np*N*sizeof(uint64_t),hipMemcpyHostToDevice,stream_mem);
    return result;
}